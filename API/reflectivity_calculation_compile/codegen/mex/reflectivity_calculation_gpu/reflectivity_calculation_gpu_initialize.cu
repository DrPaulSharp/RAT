//
//  Non-Degree Granting Education License -- for use at non-degree
//  granting, nonprofit, educational organizations only. Not for
//  government, commercial, or other organizational use.
//
//  reflectivity_calculation_gpu_initialize.cu
//
//  Code generation for function 'reflectivity_calculation_gpu_initialize'
//


// Include files
#include "reflectivity_calculation_gpu_initialize.h"
#include "_coder_reflectivity_calculation_gpu_mex.h"
#include "reflectivity_calculation_gpu.h"
#include "reflectivity_calculation_gpu_data.h"
#include "rt_nonfinite.h"

// Function Definitions
void reflectivity_calculation_gpu_initialize()
{
  mex_InitInfAndNan();
  mexFunctionCreateRootTLS();
  emlrtClearAllocCountR2012b(emlrtRootTLSGlobal, false, 0U, 0);
  emlrtEnterRtStackR2012b(emlrtRootTLSGlobal);
  emlrtLicenseCheckR2012b(emlrtRootTLSGlobal, "Distrib_Computing_Toolbox", 2);
  emlrtFirstTimeR2012b(emlrtRootTLSGlobal);
  hipGetLastError();
}

// End of code generation (reflectivity_calculation_gpu_initialize.cu)
