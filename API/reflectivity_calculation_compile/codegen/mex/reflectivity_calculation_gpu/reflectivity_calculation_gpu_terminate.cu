//
//  Non-Degree Granting Education License -- for use at non-degree
//  granting, nonprofit, educational organizations only. Not for
//  government, commercial, or other organizational use.
//
//  reflectivity_calculation_gpu_terminate.cu
//
//  Code generation for function 'reflectivity_calculation_gpu_terminate'
//


// Include files
#include "reflectivity_calculation_gpu_terminate.h"
#include "_coder_reflectivity_calculation_gpu_mex.h"
#include "reflectivity_calculation_gpu.h"
#include "reflectivity_calculation_gpu_data.h"
#include "rt_nonfinite.h"

// Function Definitions
void reflectivity_calculation_gpu_atexit()
{
  mexFunctionCreateRootTLS();
  emlrtEnterRtStackR2012b(emlrtRootTLSGlobal);
  emlrtLeaveRtStackR2012b(emlrtRootTLSGlobal);
  emlrtDestroyRootTLS(&emlrtRootTLSGlobal);
  emlrtExitTimeCleanup(&emlrtContextGlobal);
}

void reflectivity_calculation_gpu_terminate()
{
  hipError_t errCode;
  errCode = hipGetLastError();
  if (errCode != hipSuccess) {
    emlrtThinCUDAError(false, emlrtRootTLSGlobal);
  }

  emlrtLeaveRtStackR2012b(emlrtRootTLSGlobal);
  emlrtDestroyRootTLS(&emlrtRootTLSGlobal);
}

// End of code generation (reflectivity_calculation_gpu_terminate.cu)
